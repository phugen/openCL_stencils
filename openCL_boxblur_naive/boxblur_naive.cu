#include "hip/hip_runtime.h"
// An naive openCL implementation of a box blur filter.

// define an image sampler that deals with image boundaries automatically
// here: use value of border pixel that is closest to out-of-bounds pixel.
__constant sampler_t sampler = CLK_NORMALIZED_COORDS_FALSE | CLK_ADDRESS_CLAMP_TO_EDGE | CLK_FILTER_NEAREST;

// OpenCL kernel program. Takes a greyscale image, represented as a 2D array
// of integer values in a range from 0 - 255, a mask size k and writes its
// results into the output image array.
__kernel void boxblur (__read_only image2d_t image,
                       __private uint8_t k,
                       __global uint8_t* output)
{
    // retrieve this work item's global work item id in x and y dimensions
    const int2 pos = {get_global_id(0), get_global_id(1)};

    // calculate new pixel value from neighbor values
    // and respect image borders by not calculating border pixels
    uint32_t sum = 0;

    for(int i = pos.x - k; i < pos.x + k+1; i++)
    {
        for(int j = pos.y - k; j < pos.y + k+1; j++)
        {
            // add mask vector position (i, j) to position of current pixel
            // read_imagef returns a 4-vector where x is the intensity value.
            sum += *read_imagef(image, sampler, pos + (int2)(i, j)).x
        }
    }

    // clamp to uint8_t value space
    uint8_t pixelValue = clamp(sum, 0, 255);

    // divide by size of mask
    pixelValue =/ (k * k);

    // write new pixel value to output image
    output[pos[0]][pos[1]] = pixelValue;


    return;
}
